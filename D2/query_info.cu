#include "share.h"

using namespace std;

void querySystemInfo()
{
    hipDeviceProp_t prop;

    int count;

    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("------General info for device %d------\n", i);
        printf("Device Name: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock Rate: %d\n", prop.clockRate);
        printf("Device copy overlap: ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("Kernel Exectuion Timeout: ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("------Memory info for device %d------\n", i);
        printf("Total global mem:  %zu\n", prop.totalGlobalMem);
        printf("Total constant Mem:  %zu\n", prop.totalConstMem);
        printf("Max mem pitch:  %zu\n", prop.memPitch);
        printf("Texture Alignment:  %zu\n", prop.textureAlignment);
        printf("   ------ MP Information for device %d ------\n", i);
        printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp:  %zu\n", prop.sharedMemPerBlock);
        printf("Registers per mp:  %d\n", prop.regsPerBlock);
        printf("Threads in warp:  %d\n", prop.warpSize);
        printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
}
