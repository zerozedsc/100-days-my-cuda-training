// filepath: d:\Coding\100-days-my-cuda-training\D7\main.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>

// Size of vectors
#define N (1 << 24)  // 16M elements
#define BLOCK_SIZE 128
#define WARP_SIZE 32

// CUDA error checking
#define CHECK_CUDA_ERROR(call) {                                             \
    hipError_t err = call;                                                  \
    if (err != hipSuccess) {                                                \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__,     \
                hipGetErrorString(err));                                    \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
}

// 1. Warp Shuffle Reduction (Optimized)
__device__ float warpReduceSum(float sum) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }
    return sum; // The final sum will be in thread 0
}

__global__ void dotProductWarpShuffle(float *a, float *b, float *c) {
    // Allocate shared memory for partial sums
    __shared__ float sharedSum[32]; // 32 warps per block (assuming 1024 threads per block)

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int warpId = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    float sum = 0.0f;

    // Compute partial sums
    for (int i = gid; i < N; i += gridDim.x * blockDim.x) {
        sum += a[i] * b[i];
    }

    // Perform warp-level reduction
    sum = warpReduceSum(sum);

    // Store the warp sum in shared memory
    if (lane == 0) {
        sharedSum[warpId] = sum;
    }
    __syncthreads();

    // First warp reduces the partial sums from shared memory
    if (warpId == 0) {
        float finalSum = (lane < blockDim.x / WARP_SIZE) ? sharedSum[lane] : 0.0f;
        finalSum = warpReduceSum(finalSum);

        // First thread of the block adds the final sum to the global memory
        if (lane == 0) {
            atomicAdd(c, finalSum);
        }
    }
}

// 2. Global Memory Reduction
__global__ void dotProductGlobal(float *a, float *b, float *c, float *temp) {
    __shared__ float sdata[BLOCK_SIZE];
    
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    float sum = 0.0f;
    for (int i = gid; i < N; i += gridDim.x * blockDim.x) {
        sum += a[i] * b[i];
    }
    
    sdata[tid] = sum;
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        temp[blockIdx.x] = sdata[0];
    }
}

// 3. Shared Memory Reduction
__global__ void dotProductShared(float *a, float *b, float *c) {
    __shared__ float sharedSum[BLOCK_SIZE];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int i = gid; i < N; i += gridDim.x * blockDim.x) {
        sum += a[i] * b[i];
    }

    sharedSum[tid] = sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedSum[tid] += sharedSum[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(c, sharedSum[0]);
    }
}

// 4. NEW: Hierarchical Reduction - Combines warp shuffle and shared memory
__global__ void dotProductHierarchical(float *a, float *b, float *c) {
    __shared__ float sharedSum[BLOCK_SIZE / WARP_SIZE]; // One element per warp
    
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = tid % WARP_SIZE;
    int warpId = tid / WARP_SIZE;
    
    // Each thread computes partial sum
    float sum = 0.0f;
    for (int i = gid; i < N; i += gridDim.x * blockDim.x) {
        sum += a[i] * b[i];
    }
    
    // First, perform warp-level reduction using warp shuffle
    sum = warpReduceSum(sum);
    
    // Store the warp result in shared memory (only the first thread in each warp)
    if (lane == 0) {
        sharedSum[warpId] = sum;
    }
    
    __syncthreads();
    
    // Second level reduction: work on shared memory
    // Now only the first warp handles the reduction
    if (warpId == 0) {
        // Load from shared memory if the data is valid
        float warpSum = (tid < BLOCK_SIZE / WARP_SIZE) ? sharedSum[lane] : 0;
        
        // Perform warp-level reduction on these sums
        warpSum = warpReduceSum(warpSum);
        
        // The final result is now in the first thread
        if (lane == 0) {
            atomicAdd(c, warpSum);
        }
    }
}

// 5. CPU Dot Product
float dotProductCPU(float *a, float *b, int n) {
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
        sum += a[i] * b[i];
    }
    return sum;
}

int main() {
    float *h_a, *h_b;
    float *d_a, *d_b, *d_c, *d_temp;
    float h_c, cpu_result;
    
    // Allocate host memory
    h_a = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));
    
    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 1.0f;
    }
    
    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_a, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_c, sizeof(float)));
    
    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice));
    
    // Determine grid dimensions
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    numBlocks = numBlocks > 1024 ? 1024 : numBlocks;

    printf("Vector size: %d\n", N);
    printf("Block size: %d\n", BLOCK_SIZE);
    printf("Number of blocks: %d\n", numBlocks);
    
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    // 1. Test CPU Reduction
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_result = dotProductCPU(h_a, h_b, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> cpu_duration = cpu_end - cpu_start;
    printf("CPU Result: %f, Time: %f ms\n", cpu_result, cpu_duration.count());

    // 2. Test Warp Shuffle Reduction
    h_c = 0.0f;
    CHECK_CUDA_ERROR(hipMemcpy(d_c, &h_c, sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(start));
    dotProductWarpShuffle<<<numBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c);
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    float warpTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&warpTime, start, stop));
    CHECK_CUDA_ERROR(hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost));
    printf("Warp Shuffle Result: %f, Time: %f ms\n", h_c, warpTime);

    // 3. Test Global Memory Reduction
    h_c = 0.0f;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_temp, numBlocks * sizeof(float)));
    CHECK_CUDA_ERROR(hipEventRecord(start));
    dotProductGlobal<<<numBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c, d_temp);
    
    // Sum the partial results from d_temp on CPU
    float* temp_result = (float*)malloc(numBlocks * sizeof(float));
    CHECK_CUDA_ERROR(hipMemcpy(temp_result, d_temp, numBlocks * sizeof(float), hipMemcpyDeviceToHost));
    
    h_c = 0.0f;
    for (int i = 0; i < numBlocks; i++) {
        h_c += temp_result[i];
    }
    free(temp_result);
    
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));

    float globalTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&globalTime, start, stop));
    printf("Global Memory Result: %f, Time: %f ms\n", h_c, globalTime);

    // 4. Test Shared Memory Reduction
    h_c = 0.0f;
    CHECK_CUDA_ERROR(hipMemcpy(d_c, &h_c, sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(start));
    dotProductShared<<<numBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c);
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));

    float sharedTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&sharedTime, start, stop));
    CHECK_CUDA_ERROR(hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost));
    printf("Shared Memory Result: %f, Time: %f ms\n", h_c, sharedTime);

    // 5. NEW: Test Hierarchical Reduction
    h_c = 0.0f;
    CHECK_CUDA_ERROR(hipMemcpy(d_c, &h_c, sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(start));
    dotProductHierarchical<<<numBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c);
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));

    float hierarchicalTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&hierarchicalTime, start, stop));
    CHECK_CUDA_ERROR(hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost));
    printf("Hierarchical Reduction Result: %f, Time: %f ms\n", h_c, hierarchicalTime);

    // Compute and print speed comparisons
    printf("\nSpeed Comparisons:\n");

    printf("CPU vs. Warp Shuffle: %.2fx\n", cpu_duration.count() / warpTime);
    printf("CPU vs. Global Memory: %.2fx\n", cpu_duration.count() / globalTime);
    printf("CPU vs. Shared Memory: %.2fx\n", cpu_duration.count() / sharedTime);
    printf("CPU vs. Hierarchical: %.2fx\n", cpu_duration.count() / hierarchicalTime);

    printf("Warp Shuffle vs. Global Memory: %.2fx\n", globalTime / warpTime);
    printf("Warp Shuffle vs. Shared Memory: %.2fx\n", sharedTime / warpTime);
    printf("Warp Shuffle vs. Hierarchical: %.2fx\n", hierarchicalTime / warpTime);

    printf("Global Memory vs. Shared Memory: %.2fx\n", globalTime / sharedTime);
    printf("Global Memory vs. Hierarchical: %.2fx\n", globalTime / hierarchicalTime);

    printf("Shared Memory vs. Hierarchical: %.2fx\n", sharedTime / hierarchicalTime);

    // Free memory
    free(h_a);
    free(h_b);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_temp);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}